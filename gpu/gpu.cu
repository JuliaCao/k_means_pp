#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <limits>
#include <functional>
#include <cstdint>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>

/*
Author: Alexander Dunn
University of California, 2018
CS 267 Final Project: Parallelizing K-means++ Initialization
GPU portion
*/

#if defined __GNUC__ || defined __APPLE__
#include <Eigen/Dense>
#else
#include <eigen3/Eigen/Dense>
#endif

using namespace std;
using namespace Eigen;

// Command line parsing and timing: from Homework 2.3 starter files common.cu
int find_option( int argc, char **argv, const char *option )
{
    for( int i = 1; i < argc; i++ )
        if( strcmp( argv[i], option ) == 0 )
            return i;
    return -1;
}

int read_int( int argc, char **argv, const char *option, int default_value )
{
    int iplace = find_option( argc, argv, option );
    if( iplace >= 0 && iplace < argc-1 )
        return atoi( argv[iplace+1] );
    return default_value;
}

double read_timer( )
{
    static bool initialized = false;
    static struct timeval start;
    struct timeval end;
    if( !initialized )
    {
        gettimeofday( &start, NULL );
        initialized = true;
    }
    gettimeofday( &end, NULL );
    return (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
}

// Serial Indexing
template<typename Rand>
int weighted_rand_index(VectorXd& W, Rand& r){
	double culmulative = W.sum() * r();
	int i = 0;
	double s = W(0);
	while (s < culmulative){
		i++;
	  s += W(i);
	}
	return i;
}

// Serial Algorithm
template<typename Rand>
void kpp_serial(int n, int k, MatrixXd &X, MatrixXd &C, Rand &r) {

	VectorXd D(n);
	for(int i  = 0 ; i < n ; i++){
		D(i) = numeric_limits<float>::max();
	}

	// The first seed is selected uniformly at random
	int index = (int)(r() * n);
	C.row(0) = X.row(index);
	for(int j = 1; j < k; j++){
			for(auto i = 0; i < n;i++){
					VectorXd c = C.row(j-1);
					VectorXd x = X.row(i);
					VectorXd tmp = c - x;
				D(i) = min(tmp.norm(), D(i));
			}

		int i = weighted_rand_index(D,r);
	C.row(j) = X.row(i);
	}
	return;
}





//////////////////////////////////////////////////////////////////////////////

// GPU Indexing

// template<typename Rand>
struct prob_reduce
{
    __host__ __device__
        tuple<float, int> operator()(const tuple<float, int>& t1, const tuple<float, int>& t2) const {
					float w1 = get<0>(t1);
					float w2 = get<0>(t2);
					int i1 = get<1>(t1);
					int i2 = get<1>(t2);

					// NEED TO PUT ACTUAL R VALUE IN!!!!!!!!!!!!!!!
					float rval = 0.2837472 * (w1 + w2);
					if (rval > w1){
						return make_tuple(w1 + w2, i2);
					}
					else{
						return make_tuple(w1 + w2, i1);
					}
        }
};

struct D_functor
{
    const VectorXd c;
    D_functor(VectorXd _c) : c(_c) {}

    __host__ __device__
        float operator()(const VectorXd& x, const float& d) const {
					VectorXd d2 = x - c;
          return min(d2.norm(), d);
        }
};

struct conv2tuples
{
	__host __device__
		tuple<float, int> operator(const float& d, const int& i){
			return make_tuple(d, i);
		}
}

// GPU Algorithm
template<typename Rand>
void kpp_gpu(int n, int k, thrust::device_vector<float> &D,
	thrust::device_vector<int> I, thrust::device_vector<VectorXd> &X,
	thrust::device_vector<VectorXd> &C, Rand &r) {

	// The first seed is selected uniformly at random
	int index = (int)(r() * n);
	C[0] = X[index];
	for(int j = 1; j < k; j++){
			thrust::transform(X.begin(), X.end(), D.begin(), D.begin(), D_functor(C[j-1]));
			thrust::transform(D.begin(), D.end(), I.begin(), I.end(), conv2tuples());
			tuple<float, int> redtuple = thrust::reduce(D.begin(), D.end(), I.begin(), I.end(), prob_reduce());
			int ix = get<1>(redtuple);
			C[j] = X[ix];
			}
	return;
}

int main( int argc, char** argv ){

	// Parsing commands and prettify-ing output
	std::string sep = "\n-----------------------------------------------------------------------------------------------------------\n";
	int n = read_int( argc, argv, "-n", 1000);
	int m = read_int( argc, argv, "-m", 2);
	int k = read_int( argc, argv, "-k", 10);

	// Initializing Data
			// Common
	random_device rd;
	uniform_real_distribution<double> zero_one(0.f, 1.f);


			// For GPU
	auto weight_rand_gpu = bind(zero_one, ref(rd));
	thrust::device_vector<VectorXd> C_gpu(k);
	thrust::device_vector<VectorXd> X_gpu(n);
	float inf = numeric_limits<float>::max();
	thrust::device_vector<float> D_gpu(n);
	thrust::fill(D_gpu.begin(), D_gpu.end(), inf);
	// thrust::device_ptr(D)
	thrust::device_vector<int> I_gpu(n);
	thrust::sequence(I_gpu.begin(), I_gpu.end());

			// For serial
	auto weight_rand_serial = bind(zero_one, ref(rd));
	MatrixXd X_serial(n, m);
	MatrixXd C_serial(k, m);

			// Populating both serial and gpu arrays
	VectorXd randarr;
	for (int i  = 0 ; i < n ; i++){
		randarr = VectorXd::Random(m);
		X_gpu[i] = randarr;
		X_serial.row(i) = randarr;
	}

	// Running GPU simulation
	cout << sep << "RUNNING KMEANS++ GPU WITH " << n << " POINTS , " << k << " CLUSTERS, AND " << m << " DIMENSIONS.\n";
	double t0 = read_timer( );
  kpp_gpu(n, k, D_gpu, I_gpu, X_gpu, C_gpu, weight_rand_gpu);
	double t1 = read_timer( ) - t0;
	cout << "THE GPU SIMULATION TOOK " << t1 << " SECONDS. \n";


	// Initializing Data
	cout << "RUNNING KMEANS++ SERIAL WITH SAME " << n << " POINTS , " << k << " CLUSTERS, AND " << m << " DIMENSIONS.\n";
	// Running serial simulation
	double t2 = read_timer( );
  kpp_serial(n, k, X_serial, C_serial, weight_rand_serial);
	double t3 = read_timer( ) - t2;
	cout << "THE SERIAL/CPU SIMULATION TOOK " << t3 << " SECONDS. \n";
	cout << "THE RESULTING SPEEDUP IS: " << t3/t1 << sep;
}
