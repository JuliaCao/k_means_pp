#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <limits>
#include <functional>
#include <cstdint>
#include <iostream>
#include <thrust/device_vector.h>

#if defined __GNUC__ || defined __APPLE__
#include <Eigen/Dense>
#else
#include <eigen3/Eigen/Dense>
#endif

using namespace std;
using namespace Eigen;

#define M 2  // num dimensions
#define K 10  // num clusters
#define N 1000  //num points

// Command line parsing: from Homework 2.3 starter files common.cu
int find_option( int argc, char **argv, const char *option )
{
    for( int i = 1; i < argc; i++ )
        if( strcmp( argv[i], option ) == 0 )
            return i;
    return -1;
}

int read_int( int argc, char **argv, const char *option, int default_value )
{
    int iplace = find_option( argc, argv, option );
    if( iplace >= 0 && iplace < argc-1 )
        return atoi( argv[iplace+1] );
    return default_value;
}


// k-means++
template<typename Rand>
int weighted_rand_index(VectorXd& W, Rand& r){
	double culmulative = W.sum() * r();
	int i = 0;
	double s = W(0);
	while (s < culmulative){
		i++;
	  s += W(i);
	}
	return i;
}

template<typename Rand>
void kpp_gpu(MatrixXd& X, MatrixXd& C, Rand& r) {

	VectorXd D(N);
	for(int i  = 0 ; i < N ; i++){
		D(i) = numeric_limits<float>::max();
	}

	// The first seed is selected uniformly at random
	int index = (int)r() * N;
	C(0) = X(index);

	for(int j = 1; j < K; j++){
   	  for(auto i = 0;i<N;i++){
      	VectorXd c = C.row(j-1);
        VectorXd x = X.row(i);
        VectorXd tmp = c - x;
    		D(i) = min(tmp.norm(),D(i));
    	}

	  int i = weighted_rand_index(D,r);
	  C(j) = X(i);
	}
	return;
}

int main( int argc, char** argv ){

	std::string sep = "\n----------------------------------------\n";
	IOFormat CleanFmt(4, 0, ", ", "\n", "[", "]");

	int n = read_int( argc, argv, "-n", 1000 );
	int m = read_int( argc, argv, "-m", 2);
	int k = read_int( argc, argv, "-k", 10);

	cout << n;
	cout << m;
	cout << k;

	random_device rd;
	// std::mt19937 e2(rd());
	uniform_real_distribution<double> dist(-1.f, 1.f);
	uniform_real_distribution<double> zero_one(0.f, 1.f);
	auto mat_rand = bind(dist, ref(rd));
	auto weight_rand = bind(zero_one, ref(rd));

	MatrixXd X = MatrixXd::Random(n, m);
	MatrixXd C(k, m);


	// generate_data(X, mat_rand);
  kpp_gpu(X, C, weight_rand);
	// output_kmeans_pp()
}
